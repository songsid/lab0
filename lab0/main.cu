#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"
#include <hip/hip_runtime.h>
#include <string>

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

// pair word invert
__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ((idx < fsize) && (input_gpu[idx] != '\n')) {
		if (idx%2==0 && (input_gpu[idx]!=' '))
		{
			if ((input_gpu[idx] != ' ') && (input_gpu[idx + 1] != ' ') && (input_gpu[idx] != '.') && (input_gpu[idx + 1] != '.'))
			{
					char tmp = input_gpu[idx];
					input_gpu[idx] = input_gpu[idx + 1];
					input_gpu[idx + 1] = tmp;
				
					if (input_gpu[idx - 1] == ' ' || !input_gpu[idx - 1])
					{
						if (input_gpu[idx] >= 'a' && input_gpu[idx] <='z')
						{
							input_gpu[idx] -= 32;
						}
						if (input_gpu[idx + 1] >= 'A' && input_gpu[idx + 1] <= 'Z')
						{
							input_gpu[idx+1] += 32;
						}
					}
					else
					{
						if (input_gpu[idx] >= 'A' && input_gpu[idx] <= 'Z')
						{
							input_gpu[idx] += 32;
						}
						if (input_gpu[idx + 1] >= 'A' && input_gpu[idx + 1] <= 'Z')
						{
							input_gpu[idx + 1] += 32;
						}
					}
			}
			else if (input_gpu[idx + 1] == ' '&&input_gpu[idx + 1] == '.')
			{
				if (input_gpu[idx] >= 'A' && input_gpu[idx] <= 'Z')
				{
					input_gpu[idx] += 32;
				}
			}
		}
		else if (idx%2 ==1 )
		{

		}
	}
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if ( !fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);
	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);
	char *input_gpu = text_smem.get_gpu_rw();
	SomeTransform<<<fsize/32, 32>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());

	return 0;
	
}
