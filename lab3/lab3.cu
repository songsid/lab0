#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int xt = blockIdx.x * blockDim.x + threadIdx.x; //target.x
	const int yt = blockIdx.y * blockDim.y + threadIdx.y; //target.y

	const int currentNum = wt*yt + xt;

	if (yt < ht && xt < wt && mask[currentNum] > 127.0f) {
		const int yb = oy + yt, xb = ox + xt;
		const int currentBG = wb*yb + xb;

		if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
			output[currentBG * 3 + 0] = target[currentNum * 3 + 0];
			output[currentBG * 3 + 1] = target[currentNum * 3 + 1];
			output[currentBG * 3 + 2] = target[currentNum * 3 + 2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int xt = blockDim.x * blockIdx.x + threadIdx.x;
	const int yt = blockDim.y * blockIdx.y + threadIdx.y;
	const int currentNum = wt * yt + xt;

	float t_sum, b_sum;

	if (yt < ht && xt < wt) {
		const int yb = oy + yt, xb = ox + xt;
		const int currentBG = wb*yb + xb;

		for (int i = 0; i < 3; i++){
			t_sum = 0;
			b_sum = 0;

			if (xt - 1 >= 0){
				b_sum += mask[(currentNum - 1)] < 127.0 ? background[(currentBG - 1) * 3 + i] : 0;
				t_sum += target[(currentNum - 1) * 3 + i];
			}
			else{
				b_sum += background[(currentBG - 1) * 3 + i];
				t_sum += target[(currentNum)* 3 + i];
			}

			if (xt + 1 < wt){
				b_sum += mask[(currentNum + 1)] < 127.0 ? background[(currentBG + 1) * 3 + i] : 0;
				t_sum += target[(currentNum + 1) * 3 + i];
			}
			else{
				b_sum += background[(currentBG + 1) * 3 + i];
				t_sum += target[(currentNum)* 3 + i];
			}

			if (yt - 1 >= 0){
				b_sum += mask[(currentNum - wt)] < 127.0 ? background[(currentBG - wb) * 3 + i] : 0;
				t_sum += target[(currentNum - wt) * 3 + i];
			}
			else{
				b_sum += background[(currentBG - wb) * 3 + i];
				t_sum += target[(currentNum)* 3 + i];
			}

			if (yt + 1 < ht){
				b_sum += mask[(currentNum + wt)] < 127.0 ? background[(currentBG + wb) * 3 + i] : 0;
				t_sum += target[(currentNum + wt) * 3 + i];
			}
			else{
				b_sum += background[(currentBG + wb) * 3 + i];
				t_sum += target[(currentNum)* 3 + i];
			}

			fixed[currentNum * 3 + i] = 4.0*target[currentNum * 3 + i] - t_sum + b_sum;
		}
	}
}

__global__ void PoissonImageCloningInteration(
	const float *fixed,
	const float *mask,
	const float *target,
	float *output,
	const int wt,
	const int ht
	)
{

	const int xt = blockIdx.x * blockDim.x + threadIdx.x; //target.x
	const int yt = blockIdx.y * blockDim.y + threadIdx.y; //target.y
	const int currentNum = wt * yt + xt;

	float neibor_sum = 0;
	if (yt < ht && xt < wt && mask[currentNum] > 127.0f){
		for (int i = 0; i < 3; i++){
			neibor_sum = 0;

			// left
			if (xt - 1 >= 0 && mask[(currentNum - 1)] > 127.0f){
				neibor_sum += target[(currentNum - 1) * 3 + i];
			}
			// right
			if (xt + 1 < wt && mask[(currentNum + 1)] > 127.0f){
				neibor_sum += target[(currentNum + 1) * 3 + i];
			}
			// up
			if (yt - 1 >= 0 && mask[(currentNum - wt)] > 127.0f){
				neibor_sum += target[(currentNum - wt) * 3 + i];
			}
			// down
			if (yt + 1 < ht && mask[(currentNum + wt)] > 127.0f){
				neibor_sum += target[(currentNum + wt) * 3 + i];
			}

			/*if (idx == 0)
			{
				output[currentNum * 3 + i] = (fixed[currentNum * 3 + i] + neibor_sum) / 4;

			}
			if (idx == 1)
			{*/
				float w = 1.414;
				output[currentNum * 3 + i] = w * (fixed[currentNum * 3 + i] + neibor_sum) / 4 + (1.0 - w) *output[currentNum * 3 + i];
			//}



		}
	}
}
void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{

	//set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf1, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf2, 3 * wt*ht*sizeof(float));

	//initialize the iteration
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	CalculateFixed << <gdim, bdim >> >(background, target, mask, fixed, wb, hb, wt, ht, oy, ox);

	hipMemcpy(buf1, target, sizeof(float) * 3 * wt * ht, hipMemcpyDeviceToDevice);

	//iterate
	int iteratorNum = 5000;
	//if (idx == 0)  iteratorNum = 10000;
	//if (idx == 1)  iteratorNum = 5000;
	for (int i = 0; i < iteratorNum; ++i){
		PoissonImageCloningInteration << <gdim, bdim >> >(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningInteration << <gdim, bdim >> >(fixed, mask, buf2, buf1, wt, ht);
	}
	//copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float) * 3, hipMemcpyDeviceToDevice);
	SimpleClone << <gdim, bdim >> >(background, buf1, mask, output, wb, hb, wt, ht, oy, ox);

	//clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);

}
