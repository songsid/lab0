#include "hip/hip_runtime.h"
#include "counting.h"
#include "SyncedMemory.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <math.h>
#include "Timer.h"

using namespace std;
__device__ __host__ int CeilDiv(int a, int b)
{
	return (a - 1) / b + 1;
}
__device__ __host__ int CeilAlign(int a, int b)
{
	return CeilDiv(a, b) * b;
}

__constant__ char* text;
__global__ void EstablishButtonTree(const char* text, int* pos, int dataSize) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < dataSize) {
		if (text[idx] == '\n') pos[idx] = 0;
		else pos[idx] = 1;
		//printf("i=%d,idx = %d, %d\n", dataSize, idx, pos[idx]);
	}
}
__global__ void EstablishLayerTree(int* tree, int * nextTree , int currentLayer, int layer_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < layer_size && idx%2 ==0 ) {
		if (tree[idx] & tree[idx + 1])
			nextTree[idx / 2] = tree[idx] + tree[idx + 1];
		else
			nextTree[idx / 2] = 0;

		//if(layer_size < 4000)printf("i=%d idx = %d,tree[%d][%d] = %d\n", layer_size, idx, currentLayer + 1, idx / 2, nextTree[idx / 2]);
	//printf("i=%d idx = %d,tree[%d][%d] = %d\n", layer_size, idx,currentLayer,idx, tree[idx]);
	}
}
__device__ int SetPosSeq(int currentTreeIdx, int currentIdx, int * treeLayerBeginSize[], int currentNum)
{

	if (currentIdx == 0){ 
		if (treeLayerBeginSize[currentTreeIdx][currentIdx]){ return 1; }
		else return 0;
	}

	if (treeLayerBeginSize[currentTreeIdx][currentIdx])
	{
		int i = 1;
		while (treeLayerBeginSize[currentTreeIdx][currentIdx-i])
		{

			i++;
			if (currentIdx - i < 0) break;
		}
		return i;
	}

	return 0;
}
__device__ int xyToIndex(int x, int y, int text_size)
{
	int result = 0;
	for (int i = 0; i<y; i++) {
		result += (int)(text_size * pow(0.5, i));
	}
	result += x;
	return result;
}

__device__ int SetPosRecur(int currentTreeIdx, int currentIdx, int * treeLayerBeginSize, bool topDown, int currentNum,int text_size)
{
	while (1) {


		if (!topDown && treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)] && currentTreeIdx == 0)
		{
			if (currentIdx % 2 == 0){//left �G + self
				currentNum += 1;
				if (!currentIdx) return currentNum;//boundary


				if (treeLayerBeginSize[xyToIndex(currentIdx - 1, currentTreeIdx, text_size)] != 0) {
					//
					currentIdx = currentIdx - 1;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, false, currentNum,text_size);
					//
				}
				return currentNum;

			}
			else if (currentIdx % 2 == 1) // right �G find parent
			{
				if (treeLayerBeginSize[xyToIndex(currentIdx / 2, currentTreeIdx + 1, text_size)] != 0) return SetPosRecur(currentTreeIdx + 1, currentIdx / 2, treeLayerBeginSize, false, currentNum, text_size);
				else {
					if (treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)]) {
						currentNum++;
					}
					return currentNum;
				}
			}

		}
		else if (!topDown && treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)] && currentTreeIdx != 0)
		{
			if (currentIdx % 2 == 0)
			{
				currentNum += treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)];
				if (!currentIdx) return currentNum;


				if (treeLayerBeginSize[xyToIndex(currentIdx - 1, currentTreeIdx, text_size)] == 0) { // topDown
					//
					currentIdx = currentIdx - 1;
					topDown = true;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx-1, treeLayerBeginSize, true, currentNum,text_size);
					//
				}
				//
				currentIdx = currentIdx - 1;
				topDown = false;
				continue;
				//return	SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, false, currentNum,text_size);
			}
			else if (currentIdx % 2 == 1)
			{
				if (treeLayerBeginSize[xyToIndex(currentIdx / 2, currentTreeIdx + 1, text_size)] != 0)
				{
					//
					currentTreeIdx += 1;
					currentIdx /= 2;
					topDown = false;
					continue;
					//return SetPosRecur(currentTreeIdx + 1, currentIdx / 2, treeLayerBeginSize, false, currentNum,text_size);
					//
				}
				else{
					currentNum += treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)];
					//
					currentIdx -= 1;
					topDown = true;

					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, true, currentNum,text_size);
					//
				}
			}
		}
		else if (topDown)
		{
			if (treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)] == 0 && currentIdx % 2 == 0){ // left topDown
				if (currentTreeIdx == 0) return currentNum;
				//
				currentTreeIdx -= 1;
				currentIdx = currentIdx * 2 + 1;
				topDown = true;
				//return SetPosRecur(currentTreeIdx - 1, currentIdx * 2 +1, treeLayerBeginSize, true, currentNum,text_size);
			}
			else if (currentIdx % 2 == 1){ // right
				if (treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)] != 0){
					currentNum += treeLayerBeginSize[xyToIndex(currentIdx, currentTreeIdx, text_size)];
					//
					currentIdx -= 1;
					topDown = true;
					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, true, currentNum,text_size);
				}
				else
				{
					if (currentTreeIdx == 0) return currentNum;
					//
					currentIdx = currentIdx * 2 + 1;
					currentTreeIdx -= 1;
					topDown = true;
					//return SetPosRecur(currentTreeIdx - 1, currentIdx * 2 +1, treeLayerBeginSize, true, currentNum,text_size);
				}
			}
		}

		return currentNum;
	}
}
__global__ void SetPosition(int  *tree, int text_size ,int * result)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//int idxTransNum = 0;

	
	//result[idx] = SetPosRecur(0, idx, tree, false, 0, text_size);
	int currentTreeIdx = 0;
	int currentIdx = idx;
	bool topDown = false;
	int currentNum = 0;
	while (1) {


		if (!topDown && tree[xyToIndex(currentIdx, currentTreeIdx, text_size)] && currentTreeIdx == 0)
		{
			if (currentIdx % 2 == 0){//left �G + self
				currentNum += 1;
				if (!currentIdx) break;//boundary


				if (tree[xyToIndex(currentIdx - 1, currentTreeIdx, text_size)] != 0) {
					//
					currentIdx = currentIdx - 1;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, false, currentNum,text_size);
					//
				}
				break;
			}
			else if (currentIdx % 2 == 1) // right �G find parent
			{
				if (tree[xyToIndex(currentIdx / 2, currentTreeIdx + 1, text_size)] != 0) {
					//
					currentIdx /= 2;
					currentTreeIdx += 1;
					topDown = false;
					continue;
					//return SetPosRecur(currentTreeIdx + 1, currentIdx / 2, treeLayerBeginSize, false, currentNum, text_size);
					//
				}
				else {
					if (tree[xyToIndex(currentIdx, currentTreeIdx, text_size)]) {
						currentNum++;
					}
					break;
				}
			}
		}
		else if (!topDown && tree[xyToIndex(currentIdx, currentTreeIdx, text_size)] && currentTreeIdx != 0)
		{
			if (currentIdx % 2 == 0)
			{
				currentNum += tree[xyToIndex(currentIdx, currentTreeIdx, text_size)];
				if (!currentIdx) break;
				if (tree[xyToIndex(currentIdx - 1, currentTreeIdx, text_size)] == 0) { // topDown
					//
					currentIdx = currentIdx - 1;
					topDown = true;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx-1, treeLayerBeginSize, true, currentNum,text_size);
					//
				}
				//
				currentIdx = currentIdx - 1;
				topDown = false;
				continue;
				//return	SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, false, currentNum,text_size);
			}
			else if (currentIdx % 2 == 1)
			{
				if (tree[xyToIndex(currentIdx / 2, currentTreeIdx + 1, text_size)] != 0)
				{
					//
					currentTreeIdx += 1;
					currentIdx /= 2;
					topDown = false;
					continue;
					//return SetPosRecur(currentTreeIdx + 1, currentIdx / 2, treeLayerBeginSize, false, currentNum,text_size);
					//
				}
				else{
					currentNum += tree[xyToIndex(currentIdx, currentTreeIdx, text_size)];
					//
					currentIdx -= 1;
					topDown = true;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, true, currentNum,text_size);
					//
				}
			}
		}
		else if (topDown)
		{
			if (tree[xyToIndex(currentIdx, currentTreeIdx, text_size)] == 0 && currentIdx % 2 == 0){ // left topDown
				if (currentTreeIdx == 0) break;
				//
				currentTreeIdx -= 1;
				currentIdx = currentIdx * 2 + 1;
				topDown = true;
				continue;
				//return SetPosRecur(currentTreeIdx - 1, currentIdx * 2 +1, treeLayerBeginSize, true, currentNum,text_size);
			}
			else if (currentIdx % 2 == 1){ // right
				if (tree[xyToIndex(currentIdx, currentTreeIdx, text_size)] != 0){
					currentNum += tree[xyToIndex(currentIdx, currentTreeIdx, text_size)];
					//
					currentIdx -= 1;
					topDown = true;
					continue;
					//return SetPosRecur(currentTreeIdx, currentIdx - 1, treeLayerBeginSize, true, currentNum,text_size);
				}
				else
				{
					if (currentTreeIdx == 0) break;
					//
					currentIdx = currentIdx * 2 + 1;
					currentTreeIdx -= 1;
					topDown = true;
					continue;
					//return SetPosRecur(currentTreeIdx - 1, currentIdx * 2 +1, treeLayerBeginSize, true, currentNum,text_size);
				}
			}
		}

		break;
	}

	result[idx] = currentNum;
	//printf("treeeeee[%d] =%d \n", idx, currentNum);
	/*
	if (idx == 0){
		if (treeLayerBeginSize[idx]){ idxTransNum=1; }
		else idxTransNum= 0;
	
	}else if (treeLayerBeginSize[idx])
	{
		int i = 1;
		while (treeLayerBeginSize[idx - i]>0)
		{
			i++;
			if (idx - i < 0) break;
		}
		idxTransNum = i;
	}
	else
	{
		idxTransNum = 0;
	}
	*/
	

}

void CountPosition(const char *text, int *pos, int text_size)
{

	int * treeArrayTmp = (int *)malloc(text_size*sizeof(int));
 	//int * treeArray;
	int * tree[10];
	int size = text_size;

	//hipMalloc((void**)&treeArray, sizeof(int)*size);
	//hipMemcpy(treeArray, treeArrayTmp, sizeof(int)*size, hipMemcpyHostToDevice);

	int block_dim = text_size / 512 + 1;
	Timer buttonTimer;
	buttonTimer.Start();
	hipMalloc((void**)&tree[0], sizeof(int)*size);
	hipMemcpy(tree[0], treeArrayTmp, sizeof(int)*size, hipMemcpyHostToDevice);
	
	EstablishButtonTree << <block_dim, 512 >> >(text, tree[0], text_size);
	hipDeviceSynchronize();

	buttonTimer.Pause();
	//printf_timer(buttonTimer);
	//hipMemcpy(treeArrayTmp, tree[0], sizeof(int)*text_size, hipMemcpyDeviceToHost);


	int totalSize = text_size;
	int treeButtomSize = 512;

	int * treeTmp;
	Timer buttonTimer2;
	buttonTimer2.Start();

	for (int i = 1; i < 10; i++)
	{
		hipMalloc((void**)&tree[i], sizeof(int)*totalSize);
		treeTmp = (int *)malloc(sizeof(int)*totalSize);
		hipMemcpy(tree[i], treeTmp, sizeof(int)*totalSize, hipMemcpyHostToDevice);
		block_dim = (totalSize) / treeButtomSize +1;

		EstablishLayerTree << <block_dim, treeButtomSize >> >(tree[i-1], tree[i], i-1, totalSize);
		hipDeviceSynchronize(); 
		if (totalSize == 1) break;
		treeButtomSize /= 2;
		totalSize /= 2;
	}
	buttonTimer2.Pause();
	//printf_timer(buttonTimer2);


	// copy all tree
	totalSize = text_size;

	//printf("textsize = %d", totalSize);
	treeButtomSize = 512;
	int * treeBigArray = (int *)malloc(sizeof(int) * text_size * 2);
	int treeBigArrayIdx = 0;

	//
	for (int i = 0; i < 10; i++)
	{
		treeTmp = (int *)malloc(sizeof(int)*totalSize);
		hipMemcpy(treeTmp, tree[i], sizeof(int)*totalSize, hipMemcpyDeviceToHost);

		for (int j = 0; j < totalSize; j++)
		{
			treeBigArray[treeBigArrayIdx] = treeTmp[j];
			treeBigArrayIdx++;
		}
		totalSize /= 2;
		hipFree(tree[i]);
	}

	//

	int * treeInOneD;
	hipMalloc((void**)&treeInOneD, sizeof(int)*text_size * 2);

	hipMemcpy(treeInOneD,treeBigArray,sizeof(int)* text_size * 2,hipMemcpyHostToDevice);

	block_dim = text_size / 512 +1 ;
	//printf("dim  = %d",block_dim);
	int * treeResult;
	hipMalloc((void**)&treeResult, sizeof(int)*text_size );
	Timer buttonTimer3;
	buttonTimer3.Start();
	
	SetPosition << <block_dim, 512 >> >(treeInOneD, text_size, treeResult);
	
	hipDeviceSynchronize();


	buttonTimer3.Pause();
	//printf_timer(buttonTimer3);
	
	//treeTmp = (int *)malloc(sizeof(int)*text_size *2);

	//hipMemcpy(treeTmp, treeInOneD, sizeof(int)*text_size * 2, hipMemcpyDeviceToHost);

	//int * treett = (int*)malloc(sizeof(int)*text_size);
	//hipMemcpy(treett, tree[0], sizeof(int)*text_size, hipMemcpyDeviceToHost);

	//char * tmpText = (char *)malloc(sizeof(char)* text_size);
	//hipMemcpy(tmpText, text, sizeof(char)*text_size, hipMemcpyDeviceToHost);

	int * treeResultTmp = (int*)malloc(sizeof(int)* text_size);
	hipMemcpy(treeResultTmp, treeResult, sizeof(int)*text_size, hipMemcpyDeviceToHost);

	
	for (int j = 0; j < 100; j++)
	{
		//printf("treetmp[%d] =%d\n", j, treeResultTmp[j]);
	}

	hipMemcpy(pos, treeResult, sizeof(char)*text_size * 4, hipMemcpyHostToDevice);
}
template<int N>
class compare {
public:
	__device__ bool operator () (int x) { return x == N; }
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead = 0;
	hipMalloc((void**)&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);

	// TODO
	auto head_end_d =
		thrust::copy_if(
		thrust::counting_iterator<int>(0),
		thrust::counting_iterator<int>(text_size),
		pos_d,
		head_d,
		compare<1>()
		);
	nhead = head_end_d - head_d;
	hipFree(buffer);
	return nhead;
}

__global__ void P3Extra(char *input_gpu, int fsize)
{
	int idx = blockIdx.x * blockDim.x * threadIdx.x;
	if ((idx < fsize) && (input_gpu[idx] != '\n')) {

		//input_gpu[idx] = input_gpu[fsize-1-idx];

		if (idx % 2 == 0 && (input_gpu[idx] != ' '))
		{
			if ((input_gpu[idx] != ' ') && (input_gpu[idx + 1] != ' ') && (input_gpu[idx] != '.') && (input_gpu[idx + 1] != '.'))
			{
				// swap
				char tmp = input_gpu[idx];
				input_gpu[idx] = input_gpu[idx + 1];
				input_gpu[idx + 1] = tmp;
				//}
				if (input_gpu[idx - 1] == ' ' || !input_gpu[idx - 1])
				{
					if (input_gpu[idx] >= 'a' && input_gpu[idx] <= 'z')
					{
						input_gpu[idx] -= 32;
					}
					if (input_gpu[idx + 1] >= 'A' && input_gpu[idx + 1] <= 'Z')
					{
						input_gpu[idx + 1] += 32;
					}
				}
				else
				{
					if (input_gpu[idx] >= 'A' && input_gpu[idx] <= 'Z')
					{
						input_gpu[idx] += 32;
					}
					if (input_gpu[idx + 1] >= 'A' && input_gpu[idx + 1] <= 'Z')
					{
						input_gpu[idx + 1] += 32;
					}
				}
			}
			else if (input_gpu[idx + 1] == ' '&&input_gpu[idx + 1] == '.')
			{
				if (input_gpu[idx] >= 'A' && input_gpu[idx] <= 'Z')
				{
					input_gpu[idx] += 32;
				}
			}
		}
		else if (idx % 2 == 1)
		{

		}
	}
}
void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	int blockdim = text_size / 512 + 1;

	char *input_gpu = (char*)malloc(sizeof(char)*text_size);


	P3Extra << < blockdim, 512 >> > (text,text_size);

	

}
