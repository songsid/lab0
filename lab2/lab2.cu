#include "hip/hip_runtime.h"
#include "lab2.h"

// Macro functions - YUV and RGB converting.
#define clip(x) ((x) > 255 ? 255 : (x) < 0 ? 0 : (int)x)
#define RGBtoY(R, G, B) clip( ( 0.299 * R) + ( 0.587 * G) + ( 0.114 * B)      )
#define RGBtoU(R, G, B) clip( (-0.169 * R) + (-0.331 * G) + ( 0.500 * B) + 128)
#define RGBtoV(R, G, B) clip( ( 0.500 * R) + (-0.419 * G) + (-0.081 * B) + 128)

static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 960;

struct Vector3D
{
	unsigned char x, y, z;
	Vector3D(unsigned char x, unsigned char y, unsigned char z)
	{
		this->x = x;
		this->y = y;
		this->z = z;
	}
};

Lab2VideoInfo tmpInfo;
struct Lab2VideoGenerator::Impl {
	int t = 0;
};



Lab2VideoGenerator::Lab2VideoGenerator() : impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

__global__ void PCG2()
{


}
__device__ int* PCGRecurOne(int x, int y , int t , int part)
{

	//printf("x = %d , y = %d ,cosx = %f\n ",x,y,cosf(x));
	int windowDis = H*H + W*W;
	int dis = x*x + y*y;
	float disMod = (float)dis / (float)windowDis;
	//printf("dixmod =%f ,%d \n", (255 * disMod), (int)(255 * disMod));
	if (part == 1)
	{
		int tmpt = 
		t < 10 ? t = 10 : t;

		float func = cosf(y / t) - cosf(x / t);
		if (func < 0.1 && func> -0.1)
		{
			int RGB[3] = { 0, 255,  255*disMod};
			//printf("1RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}

		else
		{
			int RGB[3] = { -1, -1, -1 };
			//printf("3RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}
	}
	else if (part == 2){
		//float func = cosf(y + t) + cosf(x +t);

		float tmpt = t / NFRAME;
		t > NFRAME / 2 ? tmpt = 1 : tmpt = 6;
		float func = sinf(x+t) + sinf(y+t);
		if (func < 0.1 && func> -0.1)
		{
			int RGB[3] = { 100 - 100 * tmpt , 30   , 50 -50 * tmpt };
			//printf("1RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}

		else
		{
			int RGB[3] = { -1, -1, -1 };
			//printf("3RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}
	}
	else if (part == 3)
	{
		int tMod = t%W;
		
		float func = x*x + y*y - (2 * t*tMod);
		if (func <1000 * tMod + t && func > -1000 * tMod)
		{
		//	if (t > 30)printf("func =  %d,%d,%d \n", x, y, t);

			int RGB[3] = { 255 - 20*disMod , 189 - 80*disMod, 52 + 40*disMod };
			//printf("1RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}
		else
		{
			int RGB[3] = { -1, -1, -1 };
			//printf("3RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
			return RGB;
		}
	}
	int RGB[3] = {-1,-1,-1};
	//printf("4RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
	return RGB;
}

__global__ void PCG(Lab2VideoInfo &info, uint8_t * yuv, int tt)
{
	int t = tt;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int tPCG = t;
	//PCG2 << < 1, 1 >> > ();
	int width = idx % W;
	int height = idx / W;

	int uvWidth = width / 2;
	int uvHeight = height / 2;

	int uvIdx = uvWidth + uvHeight * (W / 2);
	int modT = t < 230 ? 240 - t : 0;
	int RGB[3] = { -1, -1, -1 };
	//int RGB[3] = { 255, 0, 0 };
	int *RGBtmp;
	for (int i = 1; i <=3; i++)
	{
		
		if (i==1) RGBtmp = PCGRecurOne(width, height, modT, i);
		
		if (RGBtmp[0] == -1 || RGBtmp[1] == -1 || RGBtmp[2] == -1) continue;
		else
		{
			if (i == 2){ int * RGBTmp2 = PCGRecurOne(width, height, tPCG, i); 
			if (RGBTmp2[0] != -1)
			{
				RGBtmp[0] = RGBTmp2[0];
				RGBtmp[1] = RGBTmp2[1];
				RGBtmp[2] = RGBTmp2[2];
			}
			}
			if (i == 3){
				int * RGBTmp2 = PCGRecurOne(width, height, tPCG, i);
				if (RGBTmp2[0] != -1)
				{
					RGBtmp[0] = RGBTmp2[0];
					RGBtmp[1] = RGBTmp2[1];
					RGBtmp[2] = RGBTmp2[2];
				}
			}

		}
		for (int j = 0; j < 3; j++)
		{
			RGB[j] = RGBtmp[j];
		}
		
	}
	
	if (RGB[0] == -1 || RGB[1] == -1 || RGB[2] == -1) return;

	yuv[idx] = (int)RGBtoY(RGB[0], RGB[1], RGB[2]);
	yuv[W*H + uvIdx] = (int)RGBtoU(RGB[0], RGB[1], RGB[2]);
	yuv[W*H + W*H / 4 + uvIdx] =(int) RGBtoV(RGB[0], RGB[1], RGB[2]);

}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;


	tmpInfo = info;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	
	hipMemset(yuv,  0, W*H);
	hipMemset(yuv + W*H, 128, W*H / 2);

	int block_dim = H*W / W;
	int t= impl->t;
	//hipMemcpy(&t, &impl->t, sizeof(int), hipMemcpyHostToDevice);
	PCG << <block_dim, W >> >(tmpInfo, yuv, t);

	++(impl->t);
}